
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 16
#define	BLOCKSIZE 4



 __global__ void contar_int_naive(int *A,  int *sal,int num1,int num2)
{
  int j=threadIdx.x;
  int cant=0;
  int i;
   for(i=0;i<N-1;i++)
      if ((A[i+j*N]==num1)&&(A[i+1+j*N]==num2))
         cant++;
 
   sal[j]=cant;
}

 __global__ void contar_int_fast(int *A,  int *sal,int num1,int num2)
{
	__shared__ int cache[BLOCKSIZE];
	int tid=threadIdx.x;
	int j = blockIdx.x;
	int cacheindex=threadIdx.x;
	int cant=0;
	while (tid<N-1){	
		if ((A[tid+j*N]==num1)&&(A[tid+1+j*N]==num2))
		      cant++;
		tid+=blockDim.x;
	}
	cache[cacheindex]=cant;
	
	__syncthreads();
	int i=blockDim.x/2;
	while (i!=0){ 
		if (cacheindex<i)
			cache[cacheindex]+=cache[cacheindex+i];
		__syncthreads();
		i=i/2;
	}
	
	if (cacheindex==0)
	sal[j]=cache[0];
}



void Print_matrix(int C[], int n) {
   int i, j;

   for (i = 0; i < n; i++) {
      for (j = 0; j < n; j++)
         printf("%d ", C[i+j*n]);
      printf("\n");
   }
}  /* Print_matrix */


void contar_int(int *A, int *sal, int num1, int num2)
{  int i,j,cant=0;
    for (j=0;j<N;j++)
       for(i=0;i<N-1;i++)
            if ((A[i+j*N]==num1)&&(A[i+1+j*N]==num2))
              cant++;

 *sal=cant;
}


 
 
 
 
 
 
 int main() {

 int i,j;
 
 
  int *A = (int *) malloc( N*N*sizeof(int) );
  int salcpu;



 //rellenar matriz de caracteres en CPU
  for (j=0;j<N;j++)
    for(i=0;i<N;i++)
   {
      A[i+N*j]=rand()% 10;
     
    }
Print_matrix(A,N);
contar_int(A,&salcpu,6,3);
printf(" \n En cpu se cuentan %d secuencias %d %d ",salcpu, 6,3);




//Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
// y lo que sea necesario






//Comienzo parte GPU

  int *sal= (int *)malloc(N*sizeof(int) ); //variable para copiar resultado de gpu a cpu
//variables para gpu
  int *dev_A;
  int *dev_sal;
  int salgpu=0;

  hipMalloc((void **) &dev_A, N*N*sizeof(int) );
  hipMalloc((void **) &dev_sal, N*sizeof(int) );
  
  hipMemcpy( dev_A, A, N*N*sizeof(int) , hipMemcpyHostToDevice );

  
  contar_int_naive<<<1,N>>>(dev_A,dev_sal,6,3);

  hipMemcpy( sal, dev_sal, N*sizeof(int) , hipMemcpyDeviceToHost );
   
    for(i=0;i<N;i++)
   {
      salgpu=salgpu+sal[i];
     
    }
  
printf(" \n En gpu se cuentan %d secuencias %d %d ",salgpu, 6,3);


 contar_int_fast<<<N,BLOCKSIZE>>>(dev_A,dev_sal,6,3);
  
  hipMemcpy( sal, dev_sal, N*sizeof(int) , hipMemcpyDeviceToHost );
int salgpuv2=0;
    for(i=0;i<N;i++)
   {
      salgpuv2=salgpuv2+sal[i];
    }



printf(" \n En gpu-fast se cuentan %d secuencias %d %d ",salgpuv2, 6,3);

  free(A);
 
  }
	
	
