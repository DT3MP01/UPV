
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

__global__ void media(double *a,  double *c)
{
  int tid=threadIdx.x;
  if(tid<N-2){
    c[tid]=(a[tid]+a[tid+1]+a[tid+2])/3.0f;
  }
}
 
 
 int main() {
 double a[N], c[N];
 double *dev_a, *dev_c;
 int i;
 
 //reservar memoria en GPU
  hipMalloc((void **) &dev_a, N*sizeof(double) );
  hipMalloc((void **) &dev_c, N*sizeof(double) );
  
  //rellenar vectores en CPU
    for (i=0;i<N;i++)
    {
     a[i]=i;
    }
    
 //enviar vectores a GPU
 hipMemcpy( dev_a, a, N*sizeof(double) , hipMemcpyHostToDevice );
 hipMemcpy( dev_c, c, N*sizeof(double) , hipMemcpyHostToDevice );
 

 media<<<1,N>>>(dev_a,dev_c);
 
 //obtener el resultado a cpu
 hipMemcpy( c, dev_c, N*sizeof(double), hipMemcpyDeviceToHost );

 for (i=0;i<N-2;i++)
  printf("  %f\n",  c[i]);
  hipFree(dev_a);
  hipFree(dev_c);

  }
	
	
