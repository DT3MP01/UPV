
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 16
#define BLOCKSIZE 4






__global__ void comprobar_gpu(int *A, int *sal)
{ int j,res=1;
  int i=blockIdx.x;

  for(j=0;j<N-1;j++)
        if (A[i+j*N]!=A[i+(1+j)*N])
            res=0;
 sal[i]=res;

}


__global__ void comprobar_gpuv2(int *A, int *sal)
{ __shared__ int cache[BLOCKSIZE];
  int tid=threadIdx.x;
  int cacheindex=threadIdx.x;
 
  int res=1;
  int i=blockIdx.x;

  while (tid<N-1){
        if (A[i+tid*N]!=A[i+(1+tid)*N])
            res=0;
            tid+= blockDim.x;
   }
  cache[cacheindex]=res;
  __syncthreads(); 
    
  int j=blockDim.x/2;
  while (j!=0)
   { if (cacheindex<j && cache[cacheindex+j] == 0)
        cache[cacheindex]=0;
     __syncthreads();
   j=j/2;
   } 
  if (cacheindex==0)
     sal[i]=cache[0];

}





void comprobar_cpu(int *A, int *sal)
{  int i,j,res=1;
    for (j=0;j<N-1;j++)
       for(i=0;i<N;i++)
            if (A[i+j*N]!=A[i+(1+j)*N])
              res=0;

    
 *sal=res;
}




void Print_matrix(int C[], int n) {
   int i, j;

   for (i = 0; i < n; i++) {
      for (j = 0; j < n; j++)
         printf("%d ", C[i+j*n]);
      printf("\n");
   }
}  /* Print_matrix */




 
 int main() {

 int i,j;
 
 
  int *A = (int *) malloc( N*N*sizeof(int) );
  int salcpu;

 //rellenar matriz de numeros en CPU
  for (j=0;j<N;j++)
    for(i=0;i<N;i++)
   {
      A[i+N*j]=i;
     
    }
A[5+N*5]=8;
Print_matrix(A,N);
comprobar_cpu(A,&salcpu);
if (salcpu==1)
printf(" \n CPU:La matriz es constante por filas");
else
 printf(" \n CPU:La matriz no es constante por filas");



//Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
// y lo que sea necesario
  int *sal= (int *)malloc(N*sizeof(int) );
  int *dev_A;
  int *dev_sal; 
  int salgpu;
 hipMalloc((void **) &dev_A, N*N*sizeof(int) ) ;
 hipMalloc((void **) &dev_sal, N*sizeof(int) ) ;

hipMemcpy( dev_A, A, N*N*sizeof(int) , hipMemcpyHostToDevice );
//VERSION 1

comprobar_gpu<<<N,1>>> (dev_A, dev_sal);

hipMemcpy( sal, dev_sal, N*sizeof(int), hipMemcpyDeviceToHost );

 salgpu=1;
 for (i=0;i<N;i++)
if (sal[i]==0)
salgpu=0;
    
    
if (salgpu==1)
printf(" \n GPU:La matriz es  constante por filas");
else
 printf(" \n GPU:La matriz no es constante por filas");

//VERSION 2

comprobar_gpuv2<<<N,BLOCKSIZE>>> (dev_A, dev_sal);   

hipMemcpy( sal, dev_sal, N*sizeof(int), hipMemcpyDeviceToHost );

 salgpu=1;
 for (i=0;i<N;i++)
if (sal[i]==0)
salgpu=0;
    
    
if (salgpu==1)
printf(" \nGPUv2:La matriz es  constante por filas\n");
else
 printf(" \n GPUv2:La matriz no es constante por filas\n");

free(A);

 
  }
	
	
