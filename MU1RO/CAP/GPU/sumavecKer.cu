
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 12
#define M 8

__global__ void add(int *a, int *b, int *c)
{
  int tidx= threadIdx.x+blockIdx.x*blockDim.x;
  int tidy= threadIdx.y + blockIdx.y*blockDim.y;
  c[tidx+tidy*M]=a[tidx+tidy*M]+b[tidx+tidy*M];
 }
 

   
 int main() {

   
 int a[N*M], b[N*M], c[N*M];
 int *dev_a, *dev_b, *dev_c,i,j;
 //reservar memoria en GPU
 hipMalloc((void **) &dev_a, N*M*sizeof(int) );
 hipMalloc((void **) &dev_b, N*M*sizeof(int) );
 hipMalloc((void **) &dev_c, N*M*sizeof(int) );
 //rellenar vectores en CPU
  for (i=0;i<M;i++)
   {
   for (j=0;j<N;j++){
     a[i+j*M]=-i;
     b[i+j*M]=i*j;
     }
    }
//enviar vectores a GPU
hipMemcpy( dev_a, a, N*M*sizeof(int) , hipMemcpyHostToDevice );
hipMemcpy( dev_b, b, N*M*sizeof(int) , hipMemcpyHostToDevice );

//llamar al Kernel
dim3 block_p_grd(2,3);
dim3 thr_p_block(4,4);

 add<<<block_p_grd,thr_p_block>>>(dev_a,dev_b,dev_c);
 //obtener el resultado de vuelta en la CPU
 hipMemcpy( c, dev_c, N*M*sizeof(int), hipMemcpyDeviceToHost );
 for (i=0;i<M;i++){
    for (j=0;j<N;j++){
  printf(" %d + %d = %d\n", a[i+j*M],b[i+j*M], c[i+j*M]);
  }
  }
  
  hipFree(dev_a) ;
  hipFree(dev_b) ;
  hipFree(dev_c) ;
  }
	
	
