
#include <hip/hip_runtime.h>
#include <stdio.h>


#define M 5
#define N 6


__global__ void mediasmatrizcpu(double *A,  double *sal)
{
  int i;
  int j=threadIdx.x;
  double suma;
  if(j<N)
    {suma=0;
    for(i=0;i<M;i++)
       suma=suma+A[i+j*M];
    sal[j]=suma/double(M);
    }
 }

void Print_matrix(double C[], int m, int n) {
   int i, j;

   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++)
         printf("%f ", C[i+j*m]);
      printf("\n");
   }
}  /* Print_matrix */
 
 int main() {
  int i,j;

  double *A = (double *) malloc( N*M*sizeof(double) );
  double *sal1 = (double *) malloc( N*sizeof(double) );
  double *dev_a , *dev_sal1;
  
  hipMalloc((void **) &dev_a, N*M*sizeof(double) );
  hipMalloc((void **) &dev_sal1, N*sizeof(double) );
 
 //rellenar matriz en CPU
  for (j=0;j<N;j++)
    for(i=0;i<M;i++)
   {
      A[i+M*j]=i+j ;
    }

  Print_matrix(A,M,N);
  
 hipMemcpy( dev_a, A, N*M*sizeof(double) , hipMemcpyHostToDevice );

  mediasmatrizcpu<<<1,N>>>(dev_a,dev_sal1);

 //obtener el resultado a cpu
 hipMemcpy( sal1, dev_sal1, N*sizeof(double), hipMemcpyDeviceToHost );

   for (j=0;j<N;j++)
    printf("media columna %d = %f  \n",j,sal1[j]);

  free(A);
  free(sal1);
  hipFree(dev_sal1);
  hipFree(dev_a);

  }
	
	
