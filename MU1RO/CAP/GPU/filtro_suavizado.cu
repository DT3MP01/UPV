#include "hip/hip_runtime.h"
#include <stdio.h>


#define M 8
#define N 12


__global__ void mediasmatrizcpu(double *A,  double *c)
{
  int tidx= threadIdx.x+blockIdx.x*blockDim.x;
  int tidy= threadIdx.y + blockIdx.y*blockDim.y;
  int txm= tidx;
  int tx = tidx+1;
  int txp = tidx+2;
  
  int tym= tidy;
  int ty = tidy+1;
  int typ = tidy+2;
  
  c[tidx+tidy*(M-2)]=(a[txm+ty*M)]+a[txp+ty*M)]+a[tx+tym*M)]+a[tx+typ*M)+a[tx+ty*M])/5.0;
  
 }

void Print_matrix(double C[], int m, int n) {
   int i, j;

   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++)
         printf("%.2f ", C[i+j*m]);
      printf("\n");
   }
}  /* Print_matrix */
 
 int main() {
  int i,j;

  double *A = (double *) malloc( N*M*sizeof(double) );
  double *sal1 = (double *) malloc( (N-2)*(M-2)*sizeof(double) );
  double *dev_a , *dev_sal1;
  
  hipMalloc((void **) &dev_a, N*M*sizeof(double) );
  hipMalloc((void **) &dev_sal1, (N-2)*(M-2)*sizeof(double) );
 
 //rellenar matriz en CPU
  for (j=0;j<N;j++)
    for(i=0;i<M;i++)
   {
      A[i+M*j]=i+j ;
    }

  Print_matrix(A,M,N);
  
 hipMemcpy( dev_a, A, N*M*sizeof(double) , hipMemcpyHostToDevice );
  
    dim3 block_p_grd(2,3);
    dim3 thr_p_block(4,4);

  mediasmatrizcpu<<<block_p_grd,thr_p_block>>>(dev_a,dev_sal1);

 //obtener el resultado a cpu
 hipMemcpy( sal1, dev_sal1, (N-2)*(M-2)*sizeof(double), hipMemcpyDeviceToHost );
 
  Print_matrix(sal1,M,N);

  free(A);
  free(sal1);
  hipFree(dev_sal1);
  hipFree(dev_a);

  }
	
	
