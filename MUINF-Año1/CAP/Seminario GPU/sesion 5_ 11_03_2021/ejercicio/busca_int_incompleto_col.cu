
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 16


void Print_matrix(int C[], int n) {
   int i, j;

   for (i = 0; i < n; i++) {
      for (j = 0; j < n; j++)
         printf("%d ", C[i+j*n]);
      printf("\n");
   }
}  /* Print_matrix */


void contar_int(int *A, int *sal, int num1, int num2)
{  int i,j,cant=0;
    for (j=0;j<N;j++)
       for(i=0;i<N-1;i++)
            if ((A[i+j*N]==num1)&&(A[i+1+j*N]==num2))
              cant++;

 *sal=cant;
}

 
 int main() {

 int i,j;
 
 
  int *A = (int *) malloc( N*N*sizeof(int) );
  int salcpu;


 //rellenar matriz de caracteres en CPU
  for (j=0;j<N;j++)
    for(i=0;i<N;i++)
   {
      A[i+N*j]=rand()% 10;
     
    }
Print_matrix(A,N);
contar_int(A,&salcpu,6,3);
printf(" \n En cpu se cuentan %d secuencias %d %d ",salcpu, 6,3);




//Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
// y lo que sea necesario

//Comienzo parte GPU

  int *sal= (int *)malloc(N*sizeof(int) ); //variable para copiar resultado de gpu a cpu
//variables para gpu
  char *dev_A;
  int *dev_sal;
  

  free(A);
 
  }
	
	
