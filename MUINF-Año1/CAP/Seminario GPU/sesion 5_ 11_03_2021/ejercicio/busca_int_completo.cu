
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 16
#define BLOCKSIZE 4


void Print_matrix(int C[], int n) {
   int i, j;

   for (i = 0; i < n; i++) {
      for (j = 0; j < n; j++)
         printf("%d ", C[i+j*n]);
      printf("\n");
   }
}  /* Print_matrix */

__global__ void contar_gpu(int *A, int *sal, int num1, int num2)
{ int i,cant=0;
  int j=blockIdx.x;

  for(i=0;i<N-1;i++)
        if ((A[i+j*N]==num1)&&(A[i+1+j*N]==num2))
            cant++;

 sal[j]=cant;

}

__global__ void contarv2(int *A, int *sal, int num1, int num2)
{
__shared__ int cache[BLOCKSIZE];
  int tid=threadIdx.x;
  int j=blockIdx.x;
  int cacheindex=threadIdx.x;
  int  cant=0;
  while (tid<N-1)
  {if ((A[j+tid*N]==num1)&&(A[j+(tid+1)*N]==num2))
            cant++;
            tid+= blockDim.x;
   } 
  cache[cacheindex]=cant;
  __syncthreads();
 int i=blockDim.x/2;
  while (i!=0)
   { if (cacheindex<i)
     cache[cacheindex]+=cache[cacheindex+i];
     __syncthreads();
   i=i/2;
   } 
  if (cacheindex==0)
     sal[j]=cache[0];

 }
 
void contar_int(int *A, int *sal, int num1, int num2)
{  int i,j,cant=0;
    for (j=0;j<N-1;j++)
       for(i=0;i<N;i++)
            if ((A[i+j*N]==num1)&&(A[i+(j+1)*N]==num2))
              cant++;

 *sal=cant;
}

 
 int main() {

 int i,j;
 
 
  int *A = (int *) malloc( N*N*sizeof(int) );
  int salcpu;
  int res=0;


 //rellenar matriz de caracteres en CPU
  for (j=0;j<N;j++)
    for(i=0;i<N;i++)
   {
      A[i+N*j]=rand()% 10;
     
    }
Print_matrix(A,N);
contar_int(A,&salcpu,6,3);
printf(" \n En cpu se cuentan %d secuencias %d %d ",salcpu, 6,3);




//Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
// y lo que sea necesario

//Comienzo parte GPU

  int *sal= (int *)malloc(N*sizeof(int) ); //variable para copiar resultado de gpu a cpu
//variables para gpu
  int *dev_A;
  int *dev_sal;
 hipMalloc((void **) &dev_A, N*N*sizeof(int) ) ;
 hipMalloc((void **) &dev_sal, N*sizeof(int) ) ;

hipMemcpy( dev_A, A, N*N*sizeof(int) , hipMemcpyHostToDevice );
contar_gpu<<<N,1>>> (dev_A, dev_sal,6,3);
hipMemcpy( sal, dev_sal, N*sizeof(int), hipMemcpyDeviceToHost );

 for (i=0;i<N;i++)
    res=res+sal[i];
   printf("\n en gpu v1  %d",res);
 
contarv2<<<N,BLOCKSIZE>>> (dev_A, dev_sal,6,3);   
 res=0;
 for (i=0;i<N;i++)
    res=res+sal[i];
   printf("\n en gpu v2  %d",res);
   
  free(A);
 
  }
	
	
