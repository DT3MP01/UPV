
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 16


void Print_matrix(int C[], int n) {
   int i, j;

   for (i = 0; i < n; i++) {
      for (j = 0; j < n; j++)
         printf("%d ", C[i+j*n]);
      printf("\n");
   }
}  /* Print_matrix */

__global__ void contar_gpu(int *A, int *sal, int num1, int num2)
{ int i,cant=0;
  int j=blockIdx.x;

  for(i=0;i<N-1;i++)
        if ((A[i+j*N]==num1)&&(A[i+1+j*N]==num2))
            cant++;

 sal[j]=cant;

}

void contar_int(int *A, int *sal, int num1, int num2)
{  int i,j,cant=0;
    for (j=0;j<N;j++)
       for(i=0;i<N-1;i++)
            if ((A[i+j*N]==num1)&&(A[i+1+j*N]==num2))
              cant++;

 *sal=cant;
}

 
 int main() {

 int i,j;
 
 
  int *A = (int *) malloc( N*N*sizeof(int) );
  int salcpu;
  int res=0;


 //rellenar matriz de caracteres en CPU
  for (j=0;j<N;j++)
    for(i=0;i<N;i++)
   {
      A[i+N*j]=rand()% 10;
     
    }
Print_matrix(A,N);
contar_int(A,&salcpu,6,3);
printf(" \n En cpu se cuentan %d secuencias %d %d ",salcpu, 6,3);




//Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
// y lo que sea necesario

//Comienzo parte GPU

  int *sal= (int *)malloc(N*sizeof(int) ); //variable para copiar resultado de gpu a cpu
//variables para gpu
  int *dev_A;
  int *dev_sal;
 hipMalloc((void **) &dev_A, N*N*sizeof(int) ) ;
 hipMalloc((void **) &dev_sal, N*sizeof(int) ) ;

hipMemcpy( dev_A, A, N*N*sizeof(int) , hipMemcpyHostToDevice );
contar_gpu<<<N,1>>> (dev_A, dev_sal,6,3);
hipMemcpy( sal, dev_sal, N*sizeof(int), hipMemcpyDeviceToHost );

 for (i=0;i<N;i++)
    res=res+sal[i];
   printf("\n en gpu  %d",res);
  free(A);
 
  }
	
	
