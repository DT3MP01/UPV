
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 8
#define BLOCKSIZE 4


void Print_matrix(int C[]) {
   int i, j;

   for (i = 0; i < N; i++) {
      for (j = 0; j < N; j++)
         printf("%d ", C[i+j*N]);
      printf("\n");
   }
}  /* Print_matrix */


void calcula_veces(int *A, int *sal)
{  int i,j;
    int cont=0;
    for (i=0;i<N;i++)
       for(j=0;j<N-1;j++)
            if (A[i+(j+1)*N]%A[i+j*N]==0)
              cont++;


 *sal=cont;
}

 __global__ void calc_veces_gpu1(int *A,  int *sal)
{
  int i,j,cont=0; 
  int tid=threadIdx.x;
 
  for (j=0;j<N-1;j++)
    if (A[tid+(j+1)*N]%A[tid+j*N]==0)
              cont++;

    sal[tid]=cont;   
 }
 
  __global__ void calc_veces_gpu2(int *A, int *sal)
  {
__shared__ int cache[BLOCKSIZE];
  int tid=threadIdx.x;
  int j=blockIdx.x;
  int cacheindex=threadIdx.x;
  int cont=0;
  while (tid<N-1)
  {if (A[j+(tid+1)*N]%A[j+tid*N]==0)
              cont++;
            tid+= blockDim.x;
   } 
  cache[cacheindex]=cont;
  __syncthreads();
 int i=blockDim.x/2;
  while (i!=0)
   { if (cacheindex<i)
     cache[cacheindex]+=cache[cacheindex+i];
     __syncthreads();
   i=i/2;
   } 
  if (cacheindex==0)
     sal[j]=cache[0];

 }
 
 int main() {

 int i,j;
 
 
  int *A = (int *) malloc( N*N*sizeof(int) );
  int salcpu;


 //rellenar matriz de enteros en CPU
  for (i=0;i<N;i++)
    for(j=0;j<N;j++)
   {
      A[i+N*j]=1+rand()% 10;
     
    }
Print_matrix(A);
calcula_veces(A,&salcpu);
printf(" \n El numero de veces en que un numero va seguido de un multiplo suyo, calculado en cpu, es %d ",salcpu);




//Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
// y lo que sea necesario

//Comienzo parte GPU

  int *sal= (int *)malloc(N*sizeof(int) ); //variable para copiar resultado parcial de gpu a cpu
//variables para gp
  
int *dev_A = (int *) malloc( N*N*sizeof(int) );
  int *dev_sal = (int *) malloc( N*sizeof(int) );
int salgpu=0;
hipMalloc((void **) &dev_A, N*N*sizeof(int) );
  hipMalloc((void **) &dev_sal, N*sizeof(int) );
 hipMemcpy( dev_A, A, N*N*sizeof(int) ,hipMemcpyHostToDevice );
calc_veces_gpu1 <<<1,N>>>(dev_A, dev_sal);
//calc_veces_gpu2 <<<N,BLOCKSIZE>>>(dev_A, dev_sal);
 hipMemcpy( sal, dev_sal, N*sizeof(int), hipMemcpyDeviceToHost );
//Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
// y lo que sea necesario

  for (j=0;j<N;j++)
     salgpu=salgpu+sal[j];



printf(" \n El numero de veces en que un numero va seguido de un multiplo suyo, calculado en gpu, es %d ",salgpu);

  free(A);
 
  }
	
	
