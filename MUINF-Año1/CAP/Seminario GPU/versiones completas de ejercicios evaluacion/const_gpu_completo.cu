
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 16
#define BLOCKSIZE 4


void Print_matrix(int C[], int n) {
   int i, j;

   for (i = 0; i < n; i++) {
      for (j = 0; j < n; j++)
         printf("%d ", C[i+j*n]);
      printf("\n");
   }
}  /* Print_matrix */


void comprobar_cpu(int *A, int *sal)
{  int i,j,res=1;
    for (j=0;j<N-1;j++)
       for(i=0;i<N;i++)
            if (A[i+j*N]!=A[i+(1+j)*N])
              res=0;

    
 *sal=res;
}

 __global__ void const_gpu1(int *A,  int *sal)
{
  int i,j,res=1; 
  int tid=threadIdx.x;
 
  for (j=0;j<N-1;j++)
    if (A[tid+j*N]!=A[tid+(1+j)*N])
              res=0;
    sal[tid]=res;
    
 }
 
 __global__ void const_gpu2(int *A, int *sal)
{
__shared__ int cache[BLOCKSIZE];
  int tid=threadIdx.x;
  int j=blockIdx.x;
  int cacheindex=threadIdx.x;
  int res=1;
  while (tid<N-1)
  {if (A[j+tid*N]!=A[j+(1+tid)*N])
            res=0;
            tid+= blockDim.x;
   } 
  cache[cacheindex]=res;
  __syncthreads();
 int i=blockDim.x/2;
  while (i!=0)
   { if (cacheindex<i)
     cache[cacheindex]*=cache[cacheindex+i]; 
     __syncthreads();
   i=i/2;
   } 
  if (cacheindex==0)
     sal[j]=cache[0];

 }
 int main() {

 int i,j;
 
 
  int *A = (int *) malloc( N*N*sizeof(int) );
  int salcpu;
  int *sal= (int *) malloc( N*sizeof(int) );
 //rellenar matriz de numeros en CPU
  for (j=0;j<N;j++)
    for(i=0;i<N;i++)
   {
      A[i+N*j]=i;
     
    }
//A[5+N*5]=8;
Print_matrix(A,N);
comprobar_cpu(A,&salcpu);
if (salcpu==1)
printf(" \n La matriz es constante por filas");
else
 printf(" \n La matriz no es constante por filas");



//Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
// y lo que sea necesario

//Comienzo parte GPU

int *dev_A = (int *) malloc( N*N*sizeof(int) );
  int *dev_sal = (int *) malloc( N*sizeof(int) );
int salgpu=1;
hipMalloc((void **) &dev_A, N*N*sizeof(int) );
  hipMalloc((void **) &dev_sal, N*sizeof(int) );
 hipMemcpy( dev_A, A, N*N*sizeof(int) ,hipMemcpyHostToDevice );
//const_gpu1 <<<1,N>>>(dev_A, dev_sal);
const_gpu2 <<<N,BLOCKSIZE>>>(dev_A, dev_sal);
 hipMemcpy( sal, dev_sal, N*sizeof(int), hipMemcpyDeviceToHost );
//Aqui pon el código para reservar memoria, copiar matriz, llamar kernel, traer resultados,
// y lo que sea necesario

  for (j=0;j<N;j++)
{    printf("%d ",sal[j]);
    if (sal[j]==0)
     salgpu=0;
     }

//Comienzo parte GPU

if (salgpu==1)
printf(" \n La matriz es constante por filas");
else
 printf(" \n La matriz no es constante por filas");

 
  }
	
	
