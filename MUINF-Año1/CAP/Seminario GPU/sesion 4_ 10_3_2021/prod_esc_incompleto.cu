


#include <hip/hip_runtime.h>
#include <stdio.h>
#define CUDA_SAFE_CALL( call ) {                                         \
 hipError_t err = call;                                                 \
 if( hipSuccess != err ) {                                              \
   fprintf(stderr,"CUDA: error occurred in cuda routine. Exiting...\n"); \
   exit(err);                                                            \
 } }
#define	BLOCKSIZE 32


//__global__ void prod_esc_gpu...
    



float prodesc_cpu(unsigned int n, float *x, float *y)
{int j;
 float suma=0.0;
   for( j=0; j<n; j++ ) 
     suma+=x[j]*y[j];
  return suma;
}
        


int main( int argc, char *argv[] ) {
  unsigned int n;
  unsigned int j;

  n =1024000;

  float *x = (float *) malloc(   n*sizeof(float) );
  float *y = (float *) malloc(   n*sizeof(float) );
  
  for( j=0; j<n; j++ ) {
    x[ j ] = 2.0f * ( (float) rand() / RAND_MAX ) - 1.0f;
    y[ j ] = 2.0f * ( (float) rand() / RAND_MAX ) - 1.0f;
  }

  hipEvent_t start, stop;
  CUDA_SAFE_CALL( hipEventCreate(&start) );
  CUDA_SAFE_CALL( hipEventCreate(&stop) );

  printf(" x*y en CPU...\n");
  CUDA_SAFE_CALL( hipEventRecord(start, NULL) ); // Record the start event
  float res=prodesc_cpu( n, x, y );
  CUDA_SAFE_CALL( hipEventRecord(stop, NULL) );  // Record the stop event
  CUDA_SAFE_CALL( hipEventSynchronize(stop) );   // Wait for the stop event to complete
  float msecCPU = 0.0f;
  CUDA_SAFE_CALL( hipEventElapsedTime(&msecCPU, start, stop) );



  printf(" x*y en GPU...\n");
  int n_blocks = n / BLOCKSIZE ;
  float *dev_x, *dev_y;
  float *dev_sal;
  float *sal= (float *)malloc(n_blocks*sizeof(float) );


 hipMalloc((void **) &dev_x, n*sizeof(float) ) ;
 hipMalloc((void **) &dev_y, n*sizeof(float) ) ;
 hipMalloc((void **) &dev_sal, n_blocks*sizeof(float) ) ;

 CUDA_SAFE_CALL( hipEventRecord(start, NULL) ); // Record the start event
 CUDA_SAFE_CALL( hipMemcpy( dev_x, x,   n*sizeof(float), hipMemcpyHostToDevice ));
 CUDA_SAFE_CALL( hipMemcpy( dev_y, y,   n*sizeof(float), hipMemcpyHostToDevice ));

  //llamada kernel producto escalar

 CUDA_SAFE_CALL( hipMemcpy( sal, dev_sal, n_blocks *sizeof(float), hipMemcpyDeviceToHost ));

  //calculos adicionales ...obtener resultado en variable res_gpu

  CUDA_SAFE_CALL( hipEventRecord(stop, NULL) );  // Record the stop event
  CUDA_SAFE_CALL( hipEventSynchronize(stop) );   // Wait for the stop event to complete
  float msecGPU = 0.0f;
  CUDA_SAFE_CALL( hipEventElapsedTime(&msecGPU, start, stop) );
  printf("CPU time = %.2f msec.\n",msecCPU);
  printf("GPU time = %.2f msec.\n",msecGPU);


  printf("res_cpu %f \n",res);
  //printf("res_gpu %f \n",res_gpu);
 

  free(x);
  free(y);
  hipFree(dev_x) ;
  hipFree(dev_y) ;
  
  

  
}

