
#include <hip/hip_runtime.h>
#include <stdio.h>
#define M 8
#define N 12

__global__ void addm(int *a, int *b, int *c)
{
  int tidx= threadIdx.x+blockIdx.x*blockDim.x ;
  int tidy= threadIdx.y + blockIdx.y*blockDim.y;
    c[tidx+tidy*M]=a[tidx+tidy*M]+b[tidx+tidy*M];
  
 }
 
 int main() {
 int a[M*N], b[M*N], c[M*N];
 int *dev_a, *dev_b, *dev_c,i,j;
 //reservar memoria en GPU
 hipMalloc((void **) &dev_a, M*N*sizeof(int) );
 hipMalloc((void **) &dev_b, M*N*sizeof(int) );
 hipMalloc((void **) &dev_c, M*N*sizeof(int) );
 //rellenar vectores en CPU
  for (i=0;i<M;i++)
   for (j=0;j<N;j++)
   {
     a[i+j*M]=-i;
     b[i+j*M]=i*j;
    }
//enviar vectores a GPU
hipMemcpy( dev_a, a, M*N*sizeof(int) , hipMemcpyHostToDevice );
hipMemcpy( dev_b, b, M*N*sizeof(int) , hipMemcpyHostToDevice );
//cudaMemcpy( dev_c, c, N*sizeof(int) , cudaMemcpyHostToDevice );

//llamar al Kernel
dim3 block_p_grd(2,3);
dim3 thr_p_block(4,4);
 addm<<<block_p_grd,thr_p_block>>>(dev_a,dev_b,dev_c);
 //obtener el resultado de vuelta en la CPU
 hipMemcpy( c, dev_c, M*N*sizeof(int), hipMemcpyDeviceToHost );
 for (i=0;i<M;i++)
  for(j=0;j<N;j++)
  printf(" %d + %d = %d\n", a[i+j*M],b[i+j*M], c[i+j*M]);
  
  hipFree(dev_a) ;
  hipFree(dev_b) ;
  hipFree(dev_c) ;
  }
	
	
