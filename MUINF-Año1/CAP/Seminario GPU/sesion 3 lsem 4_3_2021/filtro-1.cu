
#include <hip/hip_runtime.h>
#include <stdio.h>
#define M 8
#define N 12

__global__ void filtro(double *a, double*c)
{
  int tidx= threadIdx.x; //+blockIdx.x*blockDim.x ;
  int tidy= threadIdx.y; //+ blockIdx.y*blockDim.y;
  int txm=tidx;
  int tx=tidx+1;
  int txp=tidx+2;
  int tym=tidy;
  int ty=tidy+1;
  int typ=tidy+2;
    c[tidx+tidy*(M-2)]=(a[txm+ty*M]+ a[txp+ty*M]+a[tx+tym*M]+a[tx+typ*M]+a[tidx+1+(tidy+1)*M])/5.0;
  
 }
 void Print_matrix(double C[], int m, int n) {
   int i, j;

   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++)
         printf("%.2f ", C[i+j*m]);
      printf("\n");
   }
}  /* Print_matrix */

 int main() {
 double a[M*N], c[(M-2)*(N-2)];
 double *dev_a,  *dev_c;
 int i,j;
 //reservar memoria en GPU
 hipMalloc((void **) &dev_a, M*N*sizeof(double) );

 hipMalloc((void **) &dev_c, (M-2)*(N-2)*sizeof(double) );

  for (i=0;i<M;i++)
   {for (j=0;j<N;j++)
     {a[i+j*M]=i*j;}
    }
    Print_matrix(a,M,N);
//enviar vectores a GPU
hipMemcpy( dev_a, a, M*N*sizeof(double) , hipMemcpyHostToDevice );

//cudaMemcpy( dev_c, c, N*sizeof(int) , cudaMemcpyHostToDevice );

//llamar al Kernel
dim3 block_p_grd(2,3);
dim3 thr_p_block(M-2,N-2);
 filtro<<<1,thr_p_block>>>(dev_a,dev_c);
 //obtener el resultado de vuelta en la CPU
 hipMemcpy( c, dev_c, (M-2)*(N-2)*sizeof(double), hipMemcpyDeviceToHost );
 //for (i=0;i<M;i++)
 // for(j=0;j<N;j++)
 // printf(" %f + %f = %f\n", a[i+j*M],b[i+j*M], c[i+j*M]);
 printf("\n");
      Print_matrix(c,M-2,N-2);
  hipFree(dev_a) ;
  hipFree(dev_c) ;
  }
	
	
