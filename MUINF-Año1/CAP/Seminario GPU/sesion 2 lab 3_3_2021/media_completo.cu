
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

__global__ void media(double *a,  double *c)
{
  int tid=threadIdx.x; 
  c[tid]=(a[tid]+a[tid+1]+a[tid+2])/3.0f;
 }
 
 int main() {
 double a[N], c[N];
  int i;
double *dev_a, *dev_c;
 //rellenar vectores en CPU
  for (i=0;i<N;i++)
   {
     a[i]=i*i;
    }
hipMalloc((void **) &dev_a, N*sizeof(double) );
hipMalloc((void **) &dev_c, (N-2)*sizeof(double) );
hipMemcpy( dev_a, a, N*sizeof(double) , hipMemcpyHostToDevice );
 media<<<1,N-2>>>(dev_a,dev_c);
 //media(a,c);
hipMemcpy( c, dev_c, (N-2)*sizeof(double), hipMemcpyDeviceToHost );

 for (i=0;i<N-2;i++)
  printf("  %f\n",  c[i]);
  

  }
	
	
