
#include <hip/hip_runtime.h>

__global__ void ecuacion2(double *solr, double * solim, const double *a, const double *b, const double *c , int N)
{
 int id=threadIdx.x+blockIdx.x*blockDim.x;
if (id<N)
{
double tmp=b[id]*b[id]-4*a[id]*c[id];
 if (tmp>=0)
    {solr[id]=(-b[id]+sqrt(tmp))/(2*a[id]);
     solim[id]=0.0;
    }
else
  {solr[id]=(-b[id]/(2*a[id]));
   solim[id]=(sqrt(-tmp))/(2*a[id]);
  }
}
}

 
 
